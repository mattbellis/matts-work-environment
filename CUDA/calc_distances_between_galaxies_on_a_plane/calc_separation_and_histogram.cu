
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
//// notes

using namespace std;

///////////////////////////////////////////////////////////////////////////////
// GPU code to calculate the bin number.
// This assumes that you have normalized your data that you want to plot to 
// lie between 0.0 and 1.0. 
// Outside this range is classified as overflow or underflow.
///////////////////////////////////////////////////////////////////////////////
__device__ int get_bin_num(float normalized_val, int nbins)
{
    // The data goes in bins number 1 to nbins
    // 0 is the underflow
    // nbins-1 is the overflow
    //
    // Remember that we have underflow (0) and overflow (nbins-1) 
    if (normalized_val>=1.0)
    {
        // If it is greater than or equal to 1.0, put it in the overflow bin
        return nbins-1;
    }
    else if (normalized_val<0.0)
    {
        return 0;
    }
    else if (normalized_val==0.0)
    {
        return 1;
    }

    // Do this calculation only if it fails the other conditionals.
    // I think this buys us a few CPU cycles.
    int ret = (int)(normalized_val*(nbins-2)) + 1;    
    return ret;
}

///////////////////////////////////////////////////////////////////////////////
// GPU code to calculate the separation between two galaxies given the 
// right ascenscion and declanation.
///////////////////////////////////////////////////////////////////////////////
//__global__ void CalcSep(float* raA, float* decA, int ngals, int nthreads, int* hist_array, float hist_lo, float hist_hi, int hist_nbins)
__global__ void CalcSep(float* raA, float* sin_decA, float* cos_decA, int ngals, int nthreads, int* hist_array, float hist_lo, float hist_hi, int hist_nbins)
{
    //does all the i's simultaneously - one for each thread 
    int ix = blockDim.x * blockIdx.x + threadIdx.x;

    // Get normalization term
    float norm = hist_hi-hist_lo;
    float norm_val = 0;
    int bin = 0;
    int hist_array_bin_block = ix*hist_nbins;
    int hist_array_bin = 0;

    float sep=0;
    float sin_dec_ix,sin_dec_ij;
    float cos_dec_ix,cos_dec_ij;
    float ra_ix, ra_ij;
    // Do the ix ``column"
    for(int ij=ix+1;ij<ngals;ij++)
    {

        sin_dec_ix = sin_decA[ix];
        sin_dec_ij = sin_decA[ij];
        cos_dec_ix = cos_decA[ix];
        cos_dec_ij = cos_decA[ij];
        ra_ix = raA[ix];
        ra_ij = raA[ij];

        sep = acos( sin_dec_ix*sin_dec_ij + cos_dec_ix*cos_dec_ij*cos(fabs(ra_ix-ra_ij)) );

        norm_val = (sep-hist_lo)/norm;
        bin = get_bin_num(norm_val,hist_nbins);
        hist_array_bin = hist_array_bin_block + bin;

        // If we passed 0 bins or -x on the command line, don't try
        // to fill the super array.
        if (hist_nbins>2)
        {
            hist_array[hist_array_bin]++;
        }

    }//loop over gals

    // Then the ngals-ix ``column"
    ix = (ngals - 1) - ix;
    for(int ij=ix+1;ij<ngals;ij++)
    {
        sin_dec_ix = sin_decA[ix];
        sin_dec_ij = sin_decA[ij];
        cos_dec_ix = cos_decA[ix];
        cos_dec_ij = cos_decA[ij];
        ra_ix = raA[ix];
        ra_ij = raA[ij];

        sep = acos( sin_dec_ix*sin_dec_ij + cos_dec_ix*cos_dec_ij*cos(fabs(ra_ix-ra_ij)) );
                
        norm_val = (sep-hist_lo)/norm;
        bin = get_bin_num(norm_val,hist_nbins);
        hist_array_bin = hist_array_bin_block + bin;
        
        // If we passed 0 bins or -x on the command line, don't try
        // to fill the super array.
        if (hist_nbins>2)
        {
            hist_array[hist_array_bin]++;
        }
    }//loop over gals
}
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//Host code
int main(int argc, char **argv)
{
    int ngals = 2000;
    int nbins = 100;
    srand(time(0));

    ///////////////////////////////////////////////////////////////////////////
    // Grab the number of galaxies from the command line *if* they have 
    // been specified.
    if (argc>1)
    {
        ngals = atoi(argv[1]);
        if (argc>2)
        {
            nbins = atoi(argv[2]);
        }
    }
    else
    {
        printf("Usage: %s <number of galaxies> <number of histogram bins>\n",\
                argv[0]);
        printf("\nDefault is 1000 galaxies and 100 bins\n\n"); 
    }
    ///////////////////////////////////////////////////////////////////////////

    ///////////////////////////////////////////////////////////////////////////
    // Define histo arrays and memory info and the like
    size_t gal_mem_needed = ngals * sizeof(float);

    // How many threads will there be?
    int nthreads = ngals/2;
    printf("nthreads: %d\n",nthreads);

    float hist_lo = 0.0;
    float hist_hi = 3.5;

    // From here on out, use the number of bins with underflow/overflow added in
    // to the calculation.
    int nbins_with_overflow = nbins + 2;
    int nbins_in_super_hist_array = nthreads*nbins_with_overflow;

    size_t hist_mem_needed = nbins_in_super_hist_array*sizeof(int);

    ///////////////////////////////////////////////////////////////////////////
    //allocate vectors in host memory
    float* h_raA = 0; 
    float* h_decA = 0;

    float* h_sin_decA = 0; 
    float* h_cos_decA = 0;

    int *h_hist_array = 0;
    int *h_hist_array_compressed = 0;

    h_raA = (float*)malloc(gal_mem_needed);
    h_decA = (float*)malloc(gal_mem_needed);

    // Allocate memory for the cos and sin of the right asenscion. This saves
    // us some time rather than recalcuating this over and over on the GPUs.
    h_cos_decA = (float*)malloc(gal_mem_needed);
    h_sin_decA = (float*)malloc(gal_mem_needed);

    h_hist_array = (int*)malloc(hist_mem_needed);
    h_hist_array_compressed = (int*)malloc(nbins_with_overflow*sizeof(int));

    if (0==h_raA || 0==h_sin_decA || 0==h_cos_decA || 0==h_hist_array || 0==h_hist_array_compressed)
    {
        printf("Couldn't allocate memory on host....\n");
        return 1;
    }

    ///////////////////////////////////////////////////////////////////////////
    //allocate vectors in device memory
    float* d_raA=0;
    //float* d_decA=0;

    float* d_sin_decA=0;
    float* d_cos_decA=0;

    int *d_hist_array;

    hipMalloc(&d_raA, gal_mem_needed);
    //cudaMalloc(&d_decA, gal_mem_needed);

    hipMalloc(&d_cos_decA, gal_mem_needed);
    hipMalloc(&d_sin_decA, gal_mem_needed);

    hipMalloc(&d_hist_array, hist_mem_needed);

    if (0==d_raA || 0==d_cos_decA || 0==d_sin_decA || 0==d_hist_array)
    {
        printf("Couldn't allocate memory on device....\n");
        return 1;
    }

    ///////////////////////////////////////////////////////////////////////////
    // Initialise input vectors.
    // Place galaxies at random coords between 0 and 1.
    ///////////////////////////////////////////////////////////////////////////
    for(int i=0;i<ngals;i++)
    {

        h_raA[i] = rand(); 
        h_decA[i] = rand();

        h_cos_decA[i] = cos(h_decA[i]);
        h_sin_decA[i] = sin(h_decA[i]);

    }

    ///////////////////////////////////////////////////////////////////////////
    // Uncomment this section if you would also like to do the calculation on
    // the CPU.
    ///////////////////////////////////////////////////////////////////////////
    /*
    //calculate separation in CPU and calculate time needed
    clock_t teststart = clock();

    float testsep=0;
    for(int i=0;i<ngals;i++){
        for(int j=i+1;j<ngals;j++){
            testsep = acos( sin(h_decA[i])*sin(h_decA[j]) + \
            cos(h_decA[i])*cos(h_decA[j])*cos(fabs(h_raA[i]-h_raA[j])) );
        }
    }
    clock_t testend = clock();
    float testelapsed = (float)(testend-teststart);
    printf("elapsed time for CPU in ms: %f", testelapsed/CLOCKS_PER_SEC*1000);
    printf("\n");
    */
    ///////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////

    ///////////////////////////////////////////////////////////////////////////
    // Zero out the super-array that will hold the histogram entries
    // for each thread.
    printf("nbins_in_super_hist_array: %d\n",nbins_in_super_hist_array);
    for (int i=0;i<nbins_in_super_hist_array;i++)
    {
        h_hist_array[i]=0.0;
    }
    ///////////////////////////////////////////////////////////////////////////

    ///////////////////////////////////////////////////////////////////////////
    // Copy vectors from host to device memory.
    hipMemcpy(d_raA,  h_raA,  gal_mem_needed, hipMemcpyHostToDevice); 
    //cudaMemcpy(d_decA, h_decA, gal_mem_needed, cudaMemcpyHostToDevice); 

    hipMemcpy(d_sin_decA, h_sin_decA, gal_mem_needed, hipMemcpyHostToDevice); 
    hipMemcpy(d_cos_decA, h_cos_decA, gal_mem_needed, hipMemcpyHostToDevice); 

    hipMemcpy(d_hist_array, h_hist_array, hist_mem_needed, hipMemcpyHostToDevice); 

    ///////////////////////////////////////////////////////////////////////////
    // Calculate our thread/grid/block sizes.
    int threadsPerBlock = 256;

    // IS THIS CALCULATION BEING DONE PROPERLY? OPTIMALLY?????
    int blocksPerGrid = (nthreads + threadsPerBlock -1) / threadsPerBlock; //???????

    // Set up the cuda timer. 
    // Ccan't use simple CPU timer since that would only time the kernel launch overhead. 
    // Need to make sure all threads have finished before stop the timer 
    // so can synchronise threads before and after kernel launch if using cpu timer? 
    // I didn't get sensible results when I've tried that though. 

    hipEvent_t cudastart, cudaend;
    hipEventCreate(&cudastart); 
    hipEventCreate(&cudaend);

    //record the start time
    hipEventRecord(cudastart,0);

    ///////////////////////////////////////////////////////////////////////////
    // Run the kernel! 
    //CalcSep<<<blocksPerGrid, threadsPerBlock>>>(d_raA, d_decA, ngals, nthreads, d_hist_array, hist_lo, hist_hi, nbins_with_overflow);
    CalcSep<<<blocksPerGrid, threadsPerBlock>>>(d_raA, d_sin_decA, d_cos_decA, ngals, nthreads, d_hist_array, hist_lo, hist_hi, nbins_with_overflow);

    // Copy the info back off the GPU to the host.
    hipMemcpy(h_hist_array, d_hist_array, hist_mem_needed, hipMemcpyDeviceToHost); 

    ///////////////////////////////////////////////////////////////////////////
    // Record the end time
    hipEventRecord(cudaend,0);
    hipEventSynchronize(cudaend);

    ///////////////////////////////////////////////////////////////////////////
    // How long did the kernel take? this gives time in ms
    float cudaelapsed=0;
    hipEventElapsedTime(&cudaelapsed, cudastart, cudaend);
    printf("elapsed time for GPU in ms: %f\n",cudaelapsed);

    ///////////////////////////////////////////////////////////////////////////
    // Collapse the super histogram array to a simple histogram array and write
    // it out to histogram_array.txt
    int sum = 0;
    int master_bin = 0;
    for (int i=0;i<nbins_in_super_hist_array;i++)
    {
        //printf("%d ",h_hist_array[i]);
        sum += h_hist_array[i];

        master_bin = i%nbins_with_overflow;

        //printf("%d\n",master_bin);
        h_hist_array_compressed[master_bin] += h_hist_array[i];
    }
    printf("\ntotal: %d\n",sum);

    FILE *outfile; 
    outfile = fopen("histogram_array.txt","w+"); /* write to file (add text to 
                                                    a file or create a file if it does not exist.*/ 
    // Print out the compressed array
    fprintf(outfile,"%f %f\n",hist_lo,hist_hi);
    for (int i=0;i<nbins_with_overflow;i++)
    {
        fprintf(outfile,"%d ",h_hist_array_compressed[i]);
    }
    fprintf(outfile,"\n");
    fclose(outfile); /*done!*/ 

    ///////////////////////////////////////////////////////////////////////////
    // Free up the device memory.
    hipEventDestroy(cudastart);
    hipEventDestroy(cudaend);

    hipFree(d_raA); 
    //cudaFree(d_decA);
    hipFree(d_sin_decA);
    hipFree(d_cos_decA);
    hipFree(d_hist_array);

    // Free up the host memory.
    free(h_raA); 
    free(h_decA); 
    free(h_sin_decA); 
    free(h_cos_decA); 
    free(h_hist_array);
    free(h_hist_array_compressed);


}
