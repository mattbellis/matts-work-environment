#include "hip/hip_runtime.h"
__global__ void kernel (int *a)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    a[idx] = idx;
}

int main()
{
    .
    .
    kernel<<<grid,block>>>(d_a);
    .
    .
}
