
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
//// notes
// based on the examples given in the CUDE programming guide
// this one makes a list of gals, one list for ra and one for dec. 
// it can then calcs the separation between gal pairs. 
// note that it's not returning anythign from the calculation! 
// just calculating how long each process takes. 

//this version uses shared memory. I'm sticking with 128 gals
//so there's no space issues, but that would be a prob with more. 
//the trick would be to split up the arrays into chunks, 
//then put the chunks into shared mem and have one threadblock
//run over each chunk


//device code
__global__ void CalcSep(float* ra, float* dec, int ngals)
{
    //does all the i's simultaneously - one for each thread 
    int ix = blockDim.x * blockIdx.x + threadIdx.x;

    float sep=0;
    // Do 1 ``column"
    for(int ij=ix+1;ij<ngals;ij++)
    {
        sep = acos( sin(dec[ix])*sin(dec[ij]) + \
                cos(dec[ix])*cos(dec[ij])*cos(fabs(ra[ix]-ra[ij])) );
    }//loop over gals

}

//device code using shared mem
__global__ void CalcSepShared( float* ra, float* dec, int ngals)
{

    //put the ra and dec arrays into shared mem
    __shared__ float raS[128];
    __shared__ float decS[128];

    for(int i=1;i<ngals;i++){
        raS[i] = ra[i];
        decS[i] = dec[i];
    }


    //does all the i's simultaneously - one for each thread 
    int ix = blockDim.x * blockIdx.x + threadIdx.x;

    float sep=0;
    // Do 1 ``column"
    for(int ij=ix+1;ij<ngals;ij++)
    {
        sep = acos( sin(decS[ix])*sin(decS[ij]) + \
                cos(decS[ix])*cos(decS[ij])*cos(fabs(raS[ix]-raS[ij])) );
    }//loop over gals

}




//Host code
int main()
{
    int ngals = 128;


    size_t sizeneededin = ngals * sizeof(float);

    //allocate vectors in host memory
    float* h_ra = (float*)malloc(sizeneededin);
    float* h_dec = (float*)malloc(sizeneededin);
    srand(time(0));

    //initailise input vectors - place galaxies at rando coords between 0 and 1
    for(int i=0;i<ngals;i++)
    {
        h_ra[i] = rand(); 
        h_dec[i] = rand();
    }

    //Calculate separation in CPU and calculate time needed
    clock_t teststart = clock();

    float testsep=0;
    for(int i=0;i<ngals;i++){
        for(int j=i+1;j<ngals;j++){
            testsep = acos( sin(h_dec[i])*sin(h_dec[j]) + cos(h_dec[i])*cos(h_dec[j])*cos(fabs(h_ra[i]-h_ra[j])) );
        }
    }
    clock_t testend = clock();
    float testelapsed = (float)(testend-teststart);
    printf("elapsed time for CPU in ms: %f", testelapsed/CLOCKS_PER_SEC*1000);
    printf("\n");


    //allocate vectors in device memory
    float* d_ra;  float* d_dec; 
    hipMalloc(&d_ra, sizeneededin);
    hipMalloc(&d_dec, sizeneededin);

    //copy vectors from host to device memory 
    hipMemcpy(d_ra, h_ra, sizeneededin, hipMemcpyHostToDevice); 
    hipMemcpy(d_dec, h_dec, sizeneededin, hipMemcpyHostToDevice); 

    //invoke kernel
    int threadsPerBlock = (ngals*ngals)/2;
    // Only need 1/2 as many threads
    //  int blocksPerGrid = (ngals/2 + threadsPerBlock -1) / threadsPerBlock; //???????
    //this is a temp thing - to test I want all threads acessing the same memory
    int blocksPerGrid = 1;

    //set up the cuda timer. 
    hipEvent_t cudastart, cudaend;
    hipEventCreate(&cudastart); 
    hipEventCreate(&cudaend);
    //record the start time
    hipEventRecord(cudastart,0);

    //run the kernel! 
    CalcSep<<<blocksPerGrid, threadsPerBlock>>>(d_ra, d_dec, ngals);

    //record the end time
    hipEventRecord(cudaend,0);
    hipEventSynchronize(cudaend);

    //how long did the kernel take? this gives time in ms
    float cudaelapsed=0;
    hipEventElapsedTime(&cudaelapsed, cudastart, cudaend);
    printf("elapsed time for GPU in ms: %f",cudaelapsed);
    printf("\n");

    //delete memory
    hipEventDestroy(cudastart);
    hipEventDestroy(cudaend);


    //////////////////////////////////////////////////
    //now, do the same thing but with shared  memory
    /////////////////////////////////////////////////


    //set up the cuda timer. 
    hipEvent_t cudastart2, cudaend2;
    hipEventCreate(&cudastart2); 
    hipEventCreate(&cudaend2);
    //record the start time
    hipEventRecord(cudastart2,0);

    //run the kernel! 
    CalcSepShared<<<blocksPerGrid, threadsPerBlock>>>(d_ra, d_dec, ngals);

    //record the end time
    hipEventRecord(cudaend2,0);
    hipEventSynchronize(cudaend2);

    //how long did the kernel take? this gives time in ms
    float cudaelapsed2=0;
    hipEventElapsedTime(&cudaelapsed2,cudastart2, cudaend2);
    printf("elapsed time for GPU using shared memory in ms: %f",cudaelapsed2);
    printf("\n");

    //delete memory
    hipEventDestroy(cudastart2);
    hipEventDestroy(cudaend2);



    //free device memory
    hipFree(d_ra); hipFree(d_dec);
    //free host memory
    free(h_ra); free(h_dec); 

printf("size of 128 floats: %f \n", float(128*sizeof(float)));


}
